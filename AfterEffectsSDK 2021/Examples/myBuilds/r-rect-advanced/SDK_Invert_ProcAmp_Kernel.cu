#include "hip/hip_runtime.h"
#ifndef SDK_INVERT_PROC_AMP
#	define SDK_INVERT_PROC_AMP

#	include "PrGPU/KernelSupport/KernelCore.h" //includes KernelWrapper.h
#	include "PrGPU/KernelSupport/KernelMemory.h"
#	include "..\..\Headers\Math_Utils.h"
#	include "Structures.h"

#	if GF_DEVICE_TARGET_DEVICE

		GF_KERNEL_FUNCTION(
			firstPassCUDAKernel,
			((const GF_PTR(float4))(src1))
			((const GF_PTR(float4))(src2))
			((GF_PTR(float4))(src3)),

			((int)(pitch1))
			((unsigned int)(width1))
			((unsigned int)(height1))
			((int)(in16f1))

			((int)(pitch2))
			((unsigned int)(width2))
			((unsigned int)(height2))
			((int)(in16f2))

			((int)(pitch3))
			((unsigned int)(width3))
			((unsigned int)(height3))
			((int)(in16f3))

			((const GF_PTR(InvertProcAmpParams))(params)),

			((uint2)(inXY)(KERNEL_XY)))
		{
			if (inXY.x < width1 && inXY.y < height1)
			{
				// Camera

				float3 camVx = { params->camVx1, params->camVx2, params->camVx3 };
				float3 camVy = { params->camVy1, params->camVy2, params->camVy3 };
				float3 camVz = { params->camVz1, params->camVz2, params->camVz3 };
				float3 camPos = { params->camPos1, params->camPos2, params->camPos3 };
				float downsample = { params->cameraWidth / (float)width1 };

				// Layers

				float3 worldPos = getPosition(
					src1, inXY, pitch1, width1, height1, in16f1, params->depthBlackIsNear, params->depthFar,
					false, camVx, camVy, camVz, camPos, params->cameraZoom, downsample
				);

				float3 normal = { 0.0f, 0.0f, 0.0f };
				if (params->normalExistToggle) {
					normal = takeXYZf4(samplePixel(src2, inXY, pitch2, width2, height2, in16f2)); //normalized normal
					normal = subf3(mulf3(normal, 2.0f), { 1.0f, 1.0f, 1.0f }); //unnormalized normal
				}

				// Draw

				float3 draw = { 0.0f, 0.0f, 0.0f };
				float shadowsMask = 1.0f;
				
				for (int i = 0; i < 10; i++) {

					// Light Settings

					bool lightToggle = params->lightToggle[i];
					if (!lightToggle) { continue; }

					float3 pos1 = { params->posX1[i] * downsample, params->posY1[i] * downsample, params->posZ1[i] * downsample };
					float3 vX1 = { params->vXx1[i] * downsample, params->vXy1[i] * downsample, params->vXz1[i] * downsample };
					float3 vY1 = { params->vYx1[i] * downsample, params->vYy1[i] * downsample, params->vYz1[i] * downsample };
					float3 vZ1 = { params->vZx1[i] * downsample, params->vZy1[i] * downsample, params->vZz1[i] * downsample };
					float3 res1 = { params->resX1[i] * downsample, params->resY1[i] * downsample, params->resZ1[i] * downsample };
					float3 scale1 = { params->scaleX1[i] * downsample, params->scaleY1[i] * downsample, params->scaleZ1[i] * downsample };

					float3 pos2 = { params->posX2[i] * downsample, params->posY2[i] * downsample, params->posZ2[i] * downsample };
					float3 vX2 = { params->vXx2[i] * downsample, params->vXy2[i] * downsample, params->vXz2[i] * downsample };
					float3 vY2 = { params->vYx2[i] * downsample, params->vYy2[i] * downsample, params->vYz2[i] * downsample };
					float3 vZ2 = { params->vZx2[i] * downsample, params->vZy2[i] * downsample, params->vZz2[i] * downsample };
					float3 res2 = { params->resX2[i] * downsample, params->resY2[i] * downsample, params->resZ2[i] * downsample };
					float3 scale2 = { params->scaleX2[i] * downsample, params->scaleY2[i] * downsample, params->scaleZ2[i] * downsample };

					// Shape

					bool featherNormalized = params->featherNormalize[i];
					float2 featherX = { params->featherX1[i] * params->featherMultiplier, params->featherX2[i] * params->featherMultiplier };
					float2 featherY = { params->featherY1[i] * params->featherMultiplier, params->featherY2[i] * params->featherMultiplier };
					float2 featherZ = { params->featherZ1[i] * params->featherMultiplier, params->featherZ2[i] * params->featherMultiplier };
					if (featherNormalized) {
						featherX = { featherX.x / 2.0f, featherX.y / 2.0f };
						featherY = { featherY.x / 2.0f, featherY.y / 2.0f };
						featherZ = { featherZ.x / 2.0f, featherZ.y / 2.0f };
					}
					float falloff = params->falloff[i] * params->falloffMultiplier;

					// Shadow

					float shadowSampleStep = fmaxf(params->shadowSampleStep[i], 0.2f);
					float3 shadowColor = { params->shadowColorR[i], params->shadowColorG[i], params->shadowColorB[i] };

					// Draw

					float3 drawRect = { 0.0f, 0.0f, 0.0f };
					float shadows = 1.0f;
					float3 shadowsColored = { 0.0f, 0.0f, 0.0f };

					drawRect = rectAdvanced(
						// Inputs
						params->renderMode, camPos, worldPos, normal,
					 
						// Light Start
						pos1, vX1, vY1, vZ1, res1, scale1,
						// Light End
						pos2, vX2, vY2, vZ2, res2, scale2,
						// Shape
						featherX, featherY, featherZ, featherNormalized, falloff,

						// Ambient
						params->ambientToggle[i],
						params->ambientIntensity[i] * params->intensityMultiplier,
						params->ambientSaturation[i] * params->saturationMultiplier,
						{ params->ambientColorNearR[i], params->ambientColorNearG[i], params->ambientColorNearB[i] },
						params->ambientColorFarToggle[i],
						{ params->ambientColorFarR[i], params->ambientColorFarG[i], params->ambientColorFarB[i] },
						params->ambientColorFalloff[i] * params->colorFalloffMultiplier,

						// Diffuse
						params->diffuseToggle[i],
						params->diffuseIntensity[i] * params->intensityMultiplier,
						params->diffuseSaturation[i] * params->saturationMultiplier,
						{ params->diffuseColorNearR[i], params->diffuseColorNearG[i], params->diffuseColorNearB[i] },
						params->diffuseColorFarToggle[i],
						{ params->diffuseColorFarR[i], params->diffuseColorFarG[i], params->diffuseColorFarB[i] },
						params->diffuseColorFalloff[i] * params->colorFalloffMultiplier,

						// Specular
						params->specularToggle[i],
						params->specularSize[i],
						params->specularIntensity[i] * params->intensityMultiplier,
						params->specularSaturation[i] * params->saturationMultiplier,
						{ params->specularColorNearR[i], params->specularColorNearG[i], params->specularColorNearB[i] },
						params->specularColorFarToggle[i],
						{ params->specularColorFarR[i], params->specularColorFarG[i], params->specularColorFarB[i] },
						params->specularColorFalloff[i] * params->colorFalloffMultiplier,

						// Shadows Toggle
						params->shadowToggle[i], params->shadowIgnoreAmbientToggle[i], params->shadowIgnoreDiffuseToggle[i], params->shadowIgnoreSpecularToggle[i], params->shadowClipToLightToggle[i],
						// Soft Shadows
						params->shadowSoftnessRadius[i], params->shadowSoftnessSamples[i],
						// Shadows
						shadowSampleStep, params->shadowImprovedSampleRadius[i], params->shadowMaxLength[i], params->shadowThresholdStart[i], params->shadowThresholdEnd[i],
						// Shadow Visualize
						params->shadowIntensity[i], shadowColor,
						// Depth
						src1, inXY, pitch1, width1, height1, in16f1, params->depthBlackIsNear, params->depthFar,
						// Camera
						camVx, camVy, camVz, params->cameraZoom, params->cameraWidth, params->cameraHeight, downsample
					);

					if (params->renderMode == 5) {
						shadows = drawRect.x;
						shadows = mix(shadows, 1.0f, 1.0f - params->shadowIntensity[i]);
						//shadowsColored = mulf3(shadowColor, (1.0f - shadows));
						shadows = clamp(shadows, 0.0f, 1.0f);
						shadowsMask = shadowsMask * shadows;
						drawRect = { 0.0f, 0.0f, 0.0f };
					}

					draw = {
						draw.x + drawRect.x,
						draw.y + drawRect.y,
						draw.z + drawRect.z
					};

				}

				if (params->renderMode == 5) {
					draw = addf3(draw, { shadowsMask, shadowsMask, shadowsMask });
				}

				// Output

				WriteFloat4({ draw.z, draw.y, draw.x, 1.0f }, src3, inXY.y * pitch3 + inXY.x, !!in16f3);

			}
		}
#	endif

#	if __NVCC__

		void firstPassCUDA(
			float const* src1,
			float const* src2,
			float* src3,

			unsigned int pitch1,
			unsigned int width1,
			unsigned int height1,
			int	is16f1,

			unsigned int pitch2,
			unsigned int width2,
			unsigned int height2,
			int	is16f2,

			unsigned int pitch3,
			unsigned int width3,
			unsigned int height3,
			int	is16f3,

			InvertProcAmpParams* d_infoP
		)
		{
			dim3 blockDim (16, 16, 1);
			dim3 gridDim ( (width1 + blockDim.x - 1)/ blockDim.x, (height1 + blockDim.y - 1) / blockDim.y, 1 );

			firstPassCUDAKernel <<< gridDim, blockDim, 0 >>> (
				(float4 const*)src1,
				(float4 const*)src2,
				(float4*)src3,

				pitch1,
				width1,
				height1,
				is16f1,

				pitch2,
				width2,
				height2,
				is16f2,

				pitch3,
				width3,
				height3,
				is16f3,

				d_infoP
			);

			hipDeviceSynchronize();
		}

#	endif //GF_DEVICE_TARGET_HOST

#endif