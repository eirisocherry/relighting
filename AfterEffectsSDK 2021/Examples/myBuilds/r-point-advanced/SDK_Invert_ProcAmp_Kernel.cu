#include "hip/hip_runtime.h"
#ifndef SDK_INVERT_PROC_AMP
#	define SDK_INVERT_PROC_AMP

#	include "PrGPU/KernelSupport/KernelCore.h" //includes KernelWrapper.h
#	include "PrGPU/KernelSupport/KernelMemory.h"
#	include "..\..\Headers\Math_Utils.h"
#	include "Structures.h"

#	if GF_DEVICE_TARGET_DEVICE

		GF_KERNEL_FUNCTION(
			firstPassCUDAKernel,
			((const GF_PTR(float4))(src1))
			((const GF_PTR(float4))(src2))
			((GF_PTR(float4))(src3)),

			((int)(pitch1))
			((unsigned int)(width1))
			((unsigned int)(height1))
			((int)(in16f1))

			((int)(pitch2))
			((unsigned int)(width2))
			((unsigned int)(height2))
			((int)(in16f2))

			((int)(pitch3))
			((unsigned int)(width3))
			((unsigned int)(height3))
			((int)(in16f3))

			((const GF_PTR(InvertProcAmpParams))(params)),

			((uint2)(inXY)(KERNEL_XY)))
		{
			if (inXY.x < width1 && inXY.y < height1)
			{
				// Camera

				float3 camVx = { params->camVx1, params->camVx2, params->camVx3 };
				float3 camVy = { params->camVy1, params->camVy2, params->camVy3 };
				float3 camVz = { params->camVz1, params->camVz2, params->camVz3 };
				float3 camPos = { params->camPos1, params->camPos2, params->camPos3 };
				float downsample = { params->cameraWidth / (float)width1 };

				// Layers

				float3 worldPos = getPosition(
					src1, inXY, pitch1, width1, height1, in16f1, params->depthBlackIsNear, params->depthFar,
					false, camVx, camVy, camVz, camPos, params->cameraZoom, downsample
				);

				float3 normal = { 0.0f, 0.0f, 0.0f };
				if (params->normalExistToggle) {
					normal = takeXYZf4(samplePixel(src2, inXY, pitch2, width2, height2, in16f2)); //normalized normal
					normal = subf3(mulf3(normal, 2.0f), { 1.0f, 1.0f, 1.0f }); //unnormalized normal
				}

				// Draw

				float3 draw = { 0.0f, 0.0f, 0.0f };
				float shadowsMask = 1.0f;

				for (int i = 0; i < 10; i++) {

					// Light Settings

					bool lightToggle = params->lightToggle[i];
					if (!lightToggle) { continue; }

					float3 lightPos = { params->lightPosX[i] * downsample, params->lightPosY[i] * downsample, params->lightPosZ[i] * downsample };
					float radius = params->radius[i] * params->radiusMultiplier;
					float distance = distancef3(lightPos, worldPos);

					float3 drawAmbient = { 0.0f, 0.0f, 0.0f };
					float3 drawDiffuse = { 0.0f, 0.0f, 0.0f };
					float3 drawSpecular = { 0.0f, 0.0f, 0.0f };
					float shadows = 1.0f;
					float3 shadowsColored = { 0.0f, 0.0f, 0.0f };

					// Ambient, Diffuse, Specular

					if ((distance <= radius) && (params->renderMode != 5)) {

						if (params->ambientToggle[i] && ((params->renderMode == 1) || (params->renderMode == 2))) {
							drawAmbient = pointLightAmbient(
								worldPos, lightPos, radius,
								params->ambientFalloff[i],
								params->ambientIntensity[i] * params->intensityMultiplier,
								params->ambientSaturation[i] * params->saturationMultiplier,
								{ params->ambientColorNearR[i], params->ambientColorNearG[i], params->ambientColorNearB[i] },
								params->ambientColorFarToggle[i],
								{ params->ambientColorFarR[i], params->ambientColorFarG[i], params->ambientColorFarB[i] },
								params->ambientColorFalloff[i]
							);
						}

						if (params->diffuseToggle[i] && ((params->renderMode == 1) || (params->renderMode == 3))) {
							drawDiffuse = pointLightDiffuse(
								worldPos, lightPos, normal, radius,
								params->diffuseFalloff[i],
								params->diffuseIntensity[i] * params->intensityMultiplier,
								params->diffuseSaturation[i] * params->saturationMultiplier,
								{ params->diffuseColorNearR[i], params->diffuseColorNearG[i], params->diffuseColorNearB[i] },
								params->diffuseColorFarToggle[i],
								{ params->diffuseColorFarR[i], params->diffuseColorFarG[i], params->diffuseColorFarB[i] },
								params->diffuseColorFalloff[i]
							);
						}

						if (params->specularToggle[i] && ((params->renderMode == 1) || (params->renderMode == 4))) {
							drawSpecular = pointLightSpecular(
								worldPos, lightPos, normal, camPos, radius,
								params->specularSize[i],
								params->specularFalloff[i],
								params->specularIntensity[i] * params->intensityMultiplier,
								params->specularSaturation[i] * params->saturationMultiplier,
								{ params->specularColorNearR[i], params->specularColorNearG[i], params->specularColorNearB[i] },
								params->specularColorFarToggle[i],
								{ params->specularColorFarR[i], params->specularColorFarG[i], params->specularColorFarB[i] },
								params->specularColorFalloff[i]
							);
						}

					}

					// Shadows

					if (params->shadowToggle[i] && ((params->renderMode == 1) || (params->renderMode == 5))) {

						float shadowSampleStep = fmaxf(params->shadowSampleStep[i], 0.2f);
						
						if (params->shadowClipToLightToggle[i]) {
							if (distance <= radius) {
								//shadows = getSoftShadows(shadowSoftnessRadius, shadowSoftnessSamples, _PixelsWorld_uv, lightPos, shadowSampleStep, shadowStepsLimit, shadowThresholdStart, shadowThresholdEnd);
								shadows = getSoftShadows(
									params->shadowSoftnessRadius[i], params->shadowSoftnessSamples[i],
									lightPos, shadowSampleStep, params->shadowImprovedSampleRadius[i], params->shadowMaxLength[i], params->shadowThresholdStart[i], params->shadowThresholdEnd[i],
									src1, inXY, pitch1, width1, height1, in16f1, params->depthBlackIsNear, params->depthFar,
									camVx, camVy, camVz, camPos, params->cameraZoom, params->cameraWidth, params->cameraHeight, downsample
								);
							}
						}
						else {
							shadows = getSoftShadows(
								params->shadowSoftnessRadius[i], params->shadowSoftnessSamples[i],
								lightPos, shadowSampleStep, params->shadowImprovedSampleRadius[i], params->shadowMaxLength[i], params->shadowThresholdStart[i], params->shadowThresholdEnd[i],
								src1, inXY, pitch1, width1, height1, in16f1, params->depthBlackIsNear, params->depthFar,
								camVx, camVy, camVz, camPos, params->cameraZoom, params->cameraWidth, params->cameraHeight, downsample
							);
						}

						shadows = mix(shadows, 1.0f, 1.0f - params->shadowIntensity[i]);
						float3 shadowColor = { params->shadowColorR[i], params->shadowColorG[i], params->shadowColorB[i] };
						shadowsColored = mulf3(shadowColor, (1.0f - shadows));
						shadows = clamp(shadows, 0.0f, 1.0f);
						shadowsMask = shadowsMask * shadows;

						if (!params->shadowIgnoreAmbientToggle[i]) {
							drawAmbient = mulf3(drawAmbient, shadows);
						}
						if (!params->shadowIgnoreDiffuseToggle[i]) {
							drawDiffuse = mulf3(drawDiffuse, shadows);
						}
						if (!params->shadowIgnoreSpecularToggle[i]) {
							drawSpecular = mulf3(drawSpecular, shadows);
						}
						
					
					}

					draw = {
						draw.x + drawAmbient.x + drawDiffuse.x + drawSpecular.x + shadowsColored.x,
						draw.y + drawAmbient.y + drawDiffuse.y + drawSpecular.y + shadowsColored.y,
						draw.z + drawAmbient.z + drawDiffuse.z + drawSpecular.z + shadowsColored.z
					};

				}

				if (params->renderMode == 5) {
					draw = addf3(draw, { shadowsMask, shadowsMask, shadowsMask });
				}
				

				// Output

				WriteFloat4({ draw.z, draw.y, draw.x, 1.0f }, src3, inXY.y * pitch3 + inXY.x, !!in16f3);

			}
		}
#	endif

#	if __NVCC__

		void firstPassCUDA(
			float const* src1,
			float const* src2,
			float* src3,

			unsigned int pitch1,
			unsigned int width1,
			unsigned int height1,
			int	is16f1,

			unsigned int pitch2,
			unsigned int width2,
			unsigned int height2,
			int	is16f2,

			unsigned int pitch3,
			unsigned int width3,
			unsigned int height3,
			int	is16f3,

			InvertProcAmpParams* d_infoP
		)
		{
			dim3 blockDim (16, 16, 1);
			dim3 gridDim ( (width1 + blockDim.x - 1)/ blockDim.x, (height1 + blockDim.y - 1) / blockDim.y, 1 );

			firstPassCUDAKernel <<< gridDim, blockDim, 0 >>> (
				(float4 const*)src1,
				(float4 const*)src2,
				(float4*)src3,

				pitch1,
				width1,
				height1,
				is16f1,

				pitch2,
				width2,
				height2,
				is16f2,

				pitch3,
				width3,
				height3,
				is16f3,

				d_infoP
			);

			hipDeviceSynchronize();
		}

#	endif //GF_DEVICE_TARGET_HOST

#endif